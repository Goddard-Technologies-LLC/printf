#include "hip/hip_runtime.h"
// #define CATCH_CONFIG_MAIN
// #include "catch.hpp"

#include "printf_config.h"
#include "../../printf.h"

#include <string.h>
#include <iostream>
#include <sstream>
#include <memory>
#include <math.h>

// Multi-compiler-compatible local warning suppression

#if defined(_MSC_VER)
  #define DISABLE_WARNING_PUSH           __pragma(warning( push ))
  #define DISABLE_WARNING_POP            __pragma(warning( pop ))
  #define DISABLE_WARNING(warningNumber) __pragma(warning( disable : warningNumber ))

  // TODO: find the right warning number for this
  #define DISABLE_WARNING_PRINTF_FORMAT             
  #define DISABLE_WARNING_PRINTF_FORMAT_EXTRA_ARGS  
  #define DISABLE_WARNING_PRINTF_FORMAT_OVERFLOW    

#elif defined(__NVCC__)
  #define DO_PRAGMA(X) _Pragma(#X)
  #define DISABLE_WARNING_PUSH           DO_PRAGMA(push)
  #define DISABLE_WARNING_POP            DO_PRAGMA(pop)
  #define DISABLE_WARNING(warning_code)  DO_PRAGMA(diag_suppress warning_code)

  #define DISABLE_WARNING_PRINTF_FORMAT             DISABLE_WARNING(bad_printf_format_string)
  #define DISABLE_WARNING_PRINTF_FORMAT_EXTRA_ARGS 
  #define DISABLE_WARNING_PRINTF_FORMAT_OVERFLOW 

#elif defined(__GNUC__) || defined(__clang__)
  #define DO_PRAGMA(X) _Pragma(#X)
  #define DISABLE_WARNING_PUSH           DO_PRAGMA(GCC diagnostic push)
  #define DISABLE_WARNING_POP            DO_PRAGMA(GCC diagnostic pop)
  #define DISABLE_WARNING(warningName)   DO_PRAGMA(GCC diagnostic ignored #warningName)

  #define DISABLE_WARNING_PRINTF_FORMAT             DISABLE_WARNING(-Wformat)
  #define DISABLE_WARNING_PRINTF_FORMAT_EXTRA_ARGS  DISABLE_WARNING(-Wformat-extra-args)
   #define DISABLE_WARNING_PRINTF_FORMAT_OVERFLOW    DISABLE_WARNING(-Wformat-overflow)
#else
  #define DISABLE_WARNING_PUSH
  #define DISABLE_WARNING_POP
  #define DISABLE_WARNING_PRINTF_FORMAT
  #define DISABLE_WARNING_PRINTF_FORMAT_EXTRA_ARGS
  #define DISABLE_WARNING_PRINTF_FORMAT_OVERFLOW 
#endif

#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
DISABLE_WARNING_PUSH
DISABLE_WARNING_PRINTF_FORMAT
DISABLE_WARNING_PRINTF_FORMAT_EXTRA_ARGS
#endif

bool test_succeeded = true;

char* make_device_string(char const* s)
{
  size_t size = strlen(s) + 1;
  void* dsptr;
  hipMalloc(&dsptr, size);
  hipMemcpy(dsptr, s, size, hipMemcpyDefault);
  hipDeviceSynchronize();
  return (char *) dsptr;
}

inline char* mds(char const* s) { return make_device_string(s); }

struct poor_mans_string_view {
  char* data;
  size_t size;
};

struct sv_and_pos { 
	const poor_mans_string_view sv; 
	size_t pos; 
};

__device__ void append_to_buffer(char c, void* type_erased_svnp)
{ 
  auto& svnp = *(static_cast<sv_and_pos*>(type_erased_svnp));
  if (svnp.pos < svnp.sv.size) {
    svnp.sv.data[svnp.pos++] = c; 
  }
}

// output function type
typedef void (*out_fct_type)(char character, void* arg);

// ... just need to make the linker happy :-(
PRINTF_HOST void _putchar(char character)
{
  exit(EXIT_FAILURE);
}

enum class invokable {
  sprintf, vsprintf, snprintf, vsnprintf
};

__device__ __host__ char const* name(invokable inv) 
{
  switch(inv) {
  case invokable::sprintf:   return "sprintf";
  case invokable::snprintf:  return "snprintf";
  case invokable::vsprintf:  return "vsprintf";
  case invokable::vsnprintf: return "vsnprintf";
  }                     
  return "unknown";
}

__device__ int vsprintf_wrapper(char* buffer, char const* format, ...)
{
  va_list args;
  va_start(args, format);
  int ret = vsprintf_(buffer, format, args);
  va_end(args);
  return ret;
}

__device__ int vnsprintf_wrapper(char* buffer, size_t buffer_size, char const* format, ...)
{
  va_list args;
  va_start(args, format);
  int ret = vsnprintf_(buffer, buffer_size, format, args);
  va_end(args);
  return ret;
}

namespace kernels {

template <typename... Ts>
__global__ void 
invoke(
  int        * __restrict__  result, 
  invokable                  which, 
  char       * __restrict__  buffer, 
  size_t                     buffer_size, 
  char const * __restrict__  format, 
  Ts...                      args)
{
  switch(which) {
  case invokable::sprintf:   *result = sprintf_(buffer, format, args...); break;
  case invokable::snprintf:  *result = snprintf_(buffer, buffer_size, format, args...); break;
  case invokable::vsprintf:  *result = vsprintf_wrapper(buffer, format, args...); break;
  case invokable::vsnprintf: *result = vnsprintf_wrapper(buffer, buffer_size, format, args...); break;
  }
}

} // namespace kernels

template <typename... Ts>
int invoke_on_device(invokable which, char* buffer, size_t buffer_size, char const* format, Ts... args)
{
  char* buffer_d;
  char* format_d;
  int* result_d;
  int result;
  size_t format_size = strlen(format) + 1;
  hipGetLastError(); // Clearing/ignoring earlier errors
  hipMalloc(&result_d, sizeof(int));
  if (buffer != nullptr or buffer_size == 0) {
    hipMalloc(&buffer_d, buffer_size);
    hipMemcpy(buffer_d, buffer, buffer_size, hipMemcpyDefault);
  } else {
	  buffer_d = nullptr;
  }
  if (format != nullptr or format_size == 0) {
    hipMalloc(&format_d, format_size);
    hipMemcpy(format_d, format, format_size, hipMemcpyDefault);
  }
  else {
	  format_d = nullptr;
  }
  // std::cout << "Copying done, now launching kernel." << std::endl;
  kernels::invoke<<<1, 1>>>(result_d, which, buffer_d, buffer_size, format_d, args...); // Note: No perfect forwarding.
  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) { 
    throw std::runtime_error("CUDA error: " + std::string(hipGetErrorString(error))); 
  }
  if (buffer != nullptr) {
    hipMemcpy(buffer, buffer_d, buffer_size, hipMemcpyDefault);
  }
  hipMemcpy(&result, result_d, sizeof(int), hipMemcpyDefault);
  hipFree(buffer_d);
  hipFree(format_d);
  hipFree(result_d);
  hipDeviceSynchronize();
  error = hipGetLastError();
  if (error != hipSuccess) { 
    throw std::runtime_error("CUDA error: " + std::string(hipGetErrorString(error))); 
  }
  return result;
}

constexpr const size_t base_buffer_size { 100 };


template <typename... Ts>
int invoke_on_device(invokable which, char* buffer, char const* format, Ts... args)
{
  return invoke_on_device<Ts...>(which, buffer, base_buffer_size, format, args...);
}

template <typename... Ts>
int printing_check(
  const char *expected,
  const char *,
  invokable invokable_printer,
  char *buffer,
  size_t buffer_size,
  const char *format,
  Ts &&... params)
{
  if (buffer == nullptr and expected != nullptr) {
    std::cerr << "Internal error: A null buffer is expected to become non-null" << std::endl;
    exit(EXIT_FAILURE);
  }
  auto ret = invoke_on_device(invokable_printer, buffer, buffer_size, format, std::forward<Ts>(params)...);
  // std::cout << "invoked_on_device with format \"" << format << "\" done." << std::endl;
  if (buffer == nullptr) {
    return ret;
  }
  if (buffer_size != base_buffer_size) {
    buffer[base_buffer_size - 1] = '\0';
  }
  //  std::cout << "----\n";
  //  std::cout << "Resulting buffer contents: " << '"' << buffer << '"' << '\n';
  if (strncmp(buffer, expected, buffer_size) != 0) {
    buffer[strlen(expected)] = '\0';
    std::cerr << "Failed with printer " << name(invokable_printer) <<
	    " with format \"" << format << "\":\n"
		<< "Actual:   \"" << buffer   << "\"\n"
		<< "Expected: \"" << expected << "\"\n" << std::flush;
    exit(EXIT_FAILURE);
  }
  return ret;
}

template <typename... Ts>
void printing_and_ret_check(
  int expected_return_value,
  const char *expected,
  const char *,
  invokable invokable_printer,
  char *buffer,
  size_t buffer_size,
  const char *format,
  Ts &&... params)
{
    auto ret = printing_check(expected, nullptr, invokable_printer, buffer, buffer_size, format, std::forward<Ts>(params)...);
    if (ret != expected_return_value) {
      std::cerr << "Unexpected return value with printer " << name(invokable_printer) <<
      " and format \"" << format << "\":\n    Actual: " << ret << "\n    Expected: " <<
      expected_return_value << std::endl;
      exit(EXIT_FAILURE);
    }
}

namespace kernels {

__global__ void fctprintf_kernel(char* buffer)
{
  sv_and_pos svnp { {buffer, base_buffer_size}, 0 };
  fctprintf(append_to_buffer, &svnp, "This is a test of %X", 0x12EFU);
}

} // namespace kernels

void testcase_fctprintf() {
  char buffer[base_buffer_size];
  char* buffer_d;
  hipMalloc(&buffer_d, base_buffer_size);
  hipMemset(buffer_d, 0xCC, base_buffer_size);
  kernels::fctprintf_kernel<<<1, 1>>>(buffer_d);
  hipMemcpy(buffer, buffer_d, base_buffer_size, hipMemcpyDefault);
  hipDeviceSynchronize();
  if (strncmp(buffer, "This is a test of 12EF", 22U) != 0) {
    std::cerr << "fctprintf failed to produce the correct string." << std::endl;
    exit(EXIT_FAILURE);
  }
  // Remember: printf does not append a `\0` to the output after going through its format string.
  if (buffer[22] != (char)0xCC) {
    std::cerr << "fctprintf changed buffer characters past where it was allowed to\n" << std::endl;
    exit(EXIT_FAILURE);
  }
  hipFree(buffer_d);
}

PRINTF_HD static void vfctprintfbuilder_1(out_fct_type f, void* f_arg, ...)
{
  va_list args;
  va_start(args, f_arg);
  vfctprintf(f, f_arg, "This is a test of %X", args);
  va_end(args);
}

namespace kernels {

__global__ void vfctprintf(char* buffer)
{
  sv_and_pos svnp { {buffer, base_buffer_size}, 0 };
  vfctprintfbuilder_1(append_to_buffer, &svnp, 0x12EFU);
}

} // namespace kernels

void testcase_vfctprintf() {
  char buffer[base_buffer_size];
  char* buffer_d;
  hipMalloc(&buffer_d, base_buffer_size);
  hipMemset(buffer_d, 0xCC, base_buffer_size);
  kernels::vfctprintf<<<1, 1>>>(buffer_d);
  hipMemcpy(buffer, buffer_d, base_buffer_size, hipMemcpyDefault);
  hipDeviceSynchronize();
  if (strncmp(buffer, "This is a test of 12EF", 22U) != 0) {
    std::cerr << "vfctprintf failed to produce the correct string." << std::endl;
    exit(EXIT_FAILURE);
  }
  if (buffer[22] != (char)0xCC) {
    std::cerr << "vfctprintf changed buffer characters past where it was allowed to\n" << std::endl;
    exit(EXIT_FAILURE);
  }
  hipFree(buffer_d);
}

//namespace kernels {
//
//__global__ void snprintf(char* buffer, size_t buffer_size)
//{
//  snprintf_(buffer, buffer_size, "%d", -1000);
//}
//
//} // namespace kernels

void testcase_snprintf() {
  char buffer[base_buffer_size];
  printing_check("-1000", "==", invokable::snprintf, buffer, base_buffer_size, "%d", -1000);
  printing_check("-1", "==", invokable::snprintf, buffer, 3, "%d", -1000);
}

void testcase_vsprintf() {
  char buffer[base_buffer_size];
  printing_check("-1", "==", invokable::vsprintf, buffer, base_buffer_size, "%d", -1 );
  printing_check("3 -1000 test", "==", invokable::vsprintf, buffer, base_buffer_size, "%d %d %s", 3, -1000, mds("test") );
}

void testcase_vsnprintf() {
  char buffer[base_buffer_size];
  printing_check("-1", "==", invokable::vsnprintf, buffer, base_buffer_size, "%d", -1);
  printing_check("3 -1000 test", "==", invokable::vsnprintf, buffer, base_buffer_size, "%d %d %s", 3, -1000, mds("test"));
}

void testcase_simple_sprintf() {
  char buffer[base_buffer_size];
  memset(buffer, 0xCC, base_buffer_size);
  printing_check("42", "==", invokable::sprintf, buffer, base_buffer_size, "%d", 42);
}

void testcase_space_flag() {
  char buffer[base_buffer_size];
  memset(buffer, 0xCC, base_buffer_size);
  printing_check(" 42", "==", invokable::sprintf, buffer, base_buffer_size, "% d", 42);
  printing_check("-42", "==", invokable::sprintf, buffer, base_buffer_size, "% d", -42);
  printing_check("   42", "==", invokable::sprintf, buffer, base_buffer_size, "% 5d", 42);
  printing_check("  -42", "==", invokable::sprintf, buffer, base_buffer_size, "% 5d", -42);
  printing_check("             42", "==", invokable::sprintf, buffer, base_buffer_size, "% 15d", 42);
  printing_check("            -42", "==", invokable::sprintf, buffer, base_buffer_size, "% 15d", -42);
  printing_check("            -42", "==", invokable::sprintf, buffer, base_buffer_size, "% 15d", -42);
  printing_check("        -42.987", "==", invokable::sprintf, buffer, base_buffer_size, "% 15.3f", -42.987);
  printing_check("         42.987", "==", invokable::sprintf, buffer, base_buffer_size, "% 15.3f", 42.987);
  printing_check(" 1024", "==", invokable::sprintf, buffer, base_buffer_size, "% d", 1024);
  printing_check("-1024", "==", invokable::sprintf, buffer, base_buffer_size, "% d", -1024);
  printing_check(" 1024", "==", invokable::sprintf, buffer, base_buffer_size, "% i", 1024);
  printing_check("-1024", "==", invokable::sprintf, buffer, base_buffer_size, "% i", -1024);
}


#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
void testcase_space_flag__non_standard_format() {
  char buffer[base_buffer_size];
  printing_check("Hello testing", "==", invokable::sprintf, buffer, base_buffer_size, "% s", mds("Hello testing"));
  printing_check("1024", "==", invokable::sprintf, buffer, base_buffer_size, "% u", 1024);
  printing_check("4294966272", "==", invokable::sprintf, buffer, base_buffer_size, "% u", 4294966272U);
  printing_check("777", "==", invokable::sprintf, buffer, base_buffer_size, "% o", 511);
  printing_check("37777777001", "==", invokable::sprintf, buffer, base_buffer_size, "% o", 4294966785U);
  printing_check("1234abcd", "==", invokable::sprintf, buffer, base_buffer_size, "% x", 305441741);
  printing_check("edcb5433", "==", invokable::sprintf, buffer, base_buffer_size, "% x", 3989525555U);
  printing_check("1234ABCD", "==", invokable::sprintf, buffer, base_buffer_size, "% X", 305441741);
  printing_check("EDCB5433", "==", invokable::sprintf, buffer, base_buffer_size, "% X", 3989525555U);
  printing_check("x", "==", invokable::sprintf, buffer, base_buffer_size, "% c", 'x');
}
#endif

void testcase_plus_flag() {
  char buffer[base_buffer_size];
  printing_check("+42", "==", invokable::sprintf, buffer, base_buffer_size, "%+d", 42);
  printing_check("-42", "==", invokable::sprintf, buffer, base_buffer_size, "%+d", -42);
  printing_check("  +42", "==", invokable::sprintf, buffer, base_buffer_size, "%+5d", 42);
  printing_check("  -42", "==", invokable::sprintf, buffer, base_buffer_size, "%+5d", -42);
  printing_check("            +42", "==", invokable::sprintf, buffer, base_buffer_size, "%+15d", 42);
  printing_check("            -42", "==", invokable::sprintf, buffer, base_buffer_size, "%+15d", -42);
  printing_check("+1024", "==", invokable::sprintf, buffer, base_buffer_size, "%+d", 1024);
  printing_check("-1024", "==", invokable::sprintf, buffer, base_buffer_size, "%+d", -1024);
  printing_check("+1024", "==", invokable::sprintf, buffer, base_buffer_size, "%+i", 1024);
  printing_check("-1024", "==", invokable::sprintf, buffer, base_buffer_size, "%+i", -1024);
  printing_check("+", "==", invokable::sprintf, buffer, base_buffer_size, "%+.0d", 0);
}

#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
void testcase_plus_flag__non_standard_format() {
  char buffer[base_buffer_size];
  printing_check("Hello testing", "==", invokable::sprintf, buffer, base_buffer_size, "%+s", mds("Hello testing"));
  printing_check("1024", "==", invokable::sprintf, buffer, base_buffer_size, "%+u", 1024);
  printing_check("4294966272", "==", invokable::sprintf, buffer, base_buffer_size, "%+u", 4294966272U);
  printing_check("777", "==", invokable::sprintf, buffer, base_buffer_size, "%+o", 511);
  printing_check("37777777001", "==", invokable::sprintf, buffer, base_buffer_size, "%+o", 4294966785U);
  printing_check("1234abcd", "==", invokable::sprintf, buffer, base_buffer_size, "%+x", 305441741);
  printing_check("edcb5433", "==", invokable::sprintf, buffer, base_buffer_size, "%+x", 3989525555U);
  printing_check("1234ABCD", "==", invokable::sprintf, buffer, base_buffer_size, "%+X", 305441741);
  printing_check("EDCB5433", "==", invokable::sprintf, buffer, base_buffer_size, "%+X", 3989525555U);
  printing_check("x", "==", invokable::sprintf, buffer, base_buffer_size, "%+c", 'x');
}
#endif


void testcase_0_flag() {
  char buffer[base_buffer_size];
  printing_check("42", "==", invokable::sprintf, buffer, base_buffer_size, "%0d", 42);
  printing_check("42", "==", invokable::sprintf, buffer, base_buffer_size, "%0ld", 42L);
  printing_check("-42", "==", invokable::sprintf, buffer, base_buffer_size, "%0d", -42);
  printing_check("00042", "==", invokable::sprintf, buffer, base_buffer_size, "%05d", 42);
  printing_check("-0042", "==", invokable::sprintf, buffer, base_buffer_size, "%05d", -42);
  printing_check("000000000000042", "==", invokable::sprintf, buffer, base_buffer_size, "%015d", 42);
  printing_check("-00000000000042", "==", invokable::sprintf, buffer, base_buffer_size, "%015d", -42);
  printing_check("000000000042.12", "==", invokable::sprintf, buffer, base_buffer_size, "%015.2f", 42.1234);
  printing_check("00000000042.988", "==", invokable::sprintf, buffer, base_buffer_size, "%015.3f", 42.9876);
  printing_check("-00000042.98760", "==", invokable::sprintf, buffer, base_buffer_size, "%015.5f", -42.9876);
}


void testcase_minus_flag() {
  char buffer[base_buffer_size];
  printing_check("42", "==", invokable::sprintf, buffer, base_buffer_size, "%-d", 42);
  printing_check("-42", "==", invokable::sprintf, buffer, base_buffer_size, "%-d", -42);
  printing_check("42   ", "==", invokable::sprintf, buffer, base_buffer_size, "%-5d", 42);
  printing_check("-42  ", "==", invokable::sprintf, buffer, base_buffer_size, "%-5d", -42);
  printing_check("42             ", "==", invokable::sprintf, buffer, base_buffer_size, "%-15d", 42);
  printing_check("-42            ", "==", invokable::sprintf, buffer, base_buffer_size, "%-15d", -42);
}


#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
void testcase_minus_flag__non_standard_format() {
  char buffer[base_buffer_size];
  printing_check("42", "==", invokable::sprintf, buffer, base_buffer_size, "%-0d", 42);
  printing_check("-42", "==", invokable::sprintf, buffer, base_buffer_size, "%-0d", -42);
  printing_check("42   ", "==", invokable::sprintf, buffer, base_buffer_size, "%-05d", 42);
  printing_check("-42  ", "==", invokable::sprintf, buffer, base_buffer_size, "%-05d", -42);
  printing_check("42             ", "==", invokable::sprintf, buffer, base_buffer_size, "%-015d", 42);
  printing_check("-42            ", "==", invokable::sprintf, buffer, base_buffer_size, "%-015d", -42);
  printing_check("42", "==", invokable::sprintf, buffer, base_buffer_size, "%0-d", 42);
  printing_check("-42", "==", invokable::sprintf, buffer, base_buffer_size, "%0-d", -42);
  printing_check("42   ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-5d", 42);
  printing_check("-42  ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-5d", -42);
  printing_check("42             ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-15d", 42);
  printing_check("-42            ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-15d", -42);

#ifndef PRINTF_DISABLE_SUPPORT_EXPONENTIAL
  printing_check("-4.200e+01     ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-15.3e", -42.);
#else
  printing_check("e", "==", invokable::sprintf, buffer, base_buffer_size, "%0-15.3e", -42.);
#endif

#ifndef PRINTF_DISABLE_SUPPORT_EXPONENTIAL
  printing_check("-42            ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-15.3g", -42.);
#else
  printing_check("g", "==", invokable::sprintf, buffer, base_buffer_size, "%0-15.3g", -42.);
#endif
}
#endif


void testcase_hash_flag() {
  char buffer[base_buffer_size];
  printing_check("", "==", invokable::sprintf, buffer, base_buffer_size, "%#.0x", 0);
  printing_check("0", "==", invokable::sprintf, buffer, base_buffer_size, "%#.1x", 0);
  printing_check("", "==", invokable::sprintf, buffer, base_buffer_size, "%#.0llx", (long long)0);
  printing_check("0x0000614e", "==", invokable::sprintf, buffer, base_buffer_size, "%#.8x", 0x614e);
}

#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
void testcase_hash_flag__non_standard_format() {
  char buffer[base_buffer_size ];
  printing_check("0b110", "==", invokable::sprintf, buffer, base_buffer_size, "%#b", 6);
}
#endif

void testcase_specifier() {
  char buffer[base_buffer_size];

  printing_check("Hello testing", "==", invokable::sprintf, buffer, base_buffer_size, "Hello testing");
  printing_check("Hello testing", "==", invokable::sprintf, buffer, base_buffer_size, "%s", mds("Hello testing"));

DISABLE_WARNING_PUSH
DISABLE_WARNING_PRINTF_FORMAT_OVERFLOW
  printing_check("(null)", "==", invokable::sprintf, buffer, base_buffer_size, "%s", (const char*) NULL);
DISABLE_WARNING_POP
  printing_check("1024", "==", invokable::sprintf, buffer, base_buffer_size, "%d", 1024);
  printing_check("-1024", "==", invokable::sprintf, buffer, base_buffer_size, "%d", -1024);
  printing_check("1024", "==", invokable::sprintf, buffer, base_buffer_size, "%i", 1024);
  printing_check("-1024", "==", invokable::sprintf, buffer, base_buffer_size, "%i", -1024);
  printing_check("1024", "==", invokable::sprintf, buffer, base_buffer_size, "%u", 1024);
  printing_check("4294966272", "==", invokable::sprintf, buffer, base_buffer_size, "%u", 4294966272U);
  printing_check("777", "==", invokable::sprintf, buffer, base_buffer_size, "%o", 511);
  printing_check("37777777001", "==", invokable::sprintf, buffer, base_buffer_size, "%o", 4294966785U);
  printing_check("1234abcd", "==", invokable::sprintf, buffer, base_buffer_size, "%x", 305441741);
  printing_check("edcb5433", "==", invokable::sprintf, buffer, base_buffer_size, "%x", 3989525555U);
  printing_check("1234ABCD", "==", invokable::sprintf, buffer, base_buffer_size, "%X", 305441741);
  printing_check("EDCB5433", "==", invokable::sprintf, buffer, base_buffer_size, "%X", 3989525555U);
  printing_check("%", "==", invokable::sprintf, buffer, base_buffer_size, "%%");
}


void testcase_width() {
  char buffer[base_buffer_size];
  printing_check("Hello testing", "==", invokable::sprintf, buffer, base_buffer_size, "%1s", mds("Hello testing"));
  printing_check("1024", "==", invokable::sprintf, buffer, base_buffer_size, "%1d", 1024);
  printing_check("-1024", "==", invokable::sprintf, buffer, base_buffer_size, "%1d", -1024);
  printing_check("1024", "==", invokable::sprintf, buffer, base_buffer_size, "%1i", 1024);
  printing_check("-1024", "==", invokable::sprintf, buffer, base_buffer_size, "%1i", -1024);
  printing_check("1024", "==", invokable::sprintf, buffer, base_buffer_size, "%1u", 1024);
  printing_check("4294966272", "==", invokable::sprintf, buffer, base_buffer_size, "%1u", 4294966272U);
  printing_check("777", "==", invokable::sprintf, buffer, base_buffer_size, "%1o", 511);
  printing_check("37777777001", "==", invokable::sprintf, buffer, base_buffer_size, "%1o", 4294966785U);
  printing_check("1234abcd", "==", invokable::sprintf, buffer, base_buffer_size, "%1x", 305441741);
  printing_check("edcb5433", "==", invokable::sprintf, buffer, base_buffer_size, "%1x", 3989525555U);
  printing_check("1234ABCD", "==", invokable::sprintf, buffer, base_buffer_size, "%1X", 305441741);
  printing_check("EDCB5433", "==", invokable::sprintf, buffer, base_buffer_size, "%1X", 3989525555U);
  printing_check("x", "==", invokable::sprintf, buffer, base_buffer_size, "%1c", 'x');
}


void testcase_width_20() {
  char buffer[base_buffer_size];
  printing_check("               Hello", "==", invokable::sprintf, buffer, base_buffer_size, "%20s", mds("Hello"));
  printing_check("                1024", "==", invokable::sprintf, buffer, base_buffer_size, "%20d", 1024);
  printing_check("               -1024", "==", invokable::sprintf, buffer, base_buffer_size, "%20d", -1024);
  printing_check("                1024", "==", invokable::sprintf, buffer, base_buffer_size, "%20i", 1024);
  printing_check("               -1024", "==", invokable::sprintf, buffer, base_buffer_size, "%20i", -1024);
  printing_check("                1024", "==", invokable::sprintf, buffer, base_buffer_size, "%20u", 1024);
  printing_check("          4294966272", "==", invokable::sprintf, buffer, base_buffer_size, "%20u", 4294966272U);
  printing_check("                 777", "==", invokable::sprintf, buffer, base_buffer_size, "%20o", 511);
  printing_check("         37777777001", "==", invokable::sprintf, buffer, base_buffer_size, "%20o", 4294966785U);
  printing_check("            1234abcd", "==", invokable::sprintf, buffer, base_buffer_size, "%20x", 305441741);
  printing_check("            edcb5433", "==", invokable::sprintf, buffer, base_buffer_size, "%20x", 3989525555U);
  printing_check("            1234ABCD", "==", invokable::sprintf, buffer, base_buffer_size, "%20X", 305441741);
  printing_check("            EDCB5433", "==", invokable::sprintf, buffer, base_buffer_size, "%20X", 3989525555U);
  printing_check("                   x", "==", invokable::sprintf, buffer, base_buffer_size, "%20c", 'x');
}


void testcase_width_star_20() {
  char buffer[base_buffer_size];
  printing_check("               Hello", "==", invokable::sprintf, buffer, base_buffer_size, "%*s", 20, mds("Hello"));
  printing_check("                1024", "==", invokable::sprintf, buffer, base_buffer_size, "%*d", 20, 1024);
  printing_check("               -1024", "==", invokable::sprintf, buffer, base_buffer_size, "%*d", 20, -1024);
  printing_check("                1024", "==", invokable::sprintf, buffer, base_buffer_size, "%*i", 20, 1024);
  printing_check("               -1024", "==", invokable::sprintf, buffer, base_buffer_size, "%*i", 20, -1024);
  printing_check("                1024", "==", invokable::sprintf, buffer, base_buffer_size, "%*u", 20, 1024);
  printing_check("          4294966272", "==", invokable::sprintf, buffer, base_buffer_size, "%*u", 20, 4294966272U);
  printing_check("                 777", "==", invokable::sprintf, buffer, base_buffer_size, "%*o", 20, 511);
  printing_check("         37777777001", "==", invokable::sprintf, buffer, base_buffer_size, "%*o", 20, 4294966785U);
  printing_check("            1234abcd", "==", invokable::sprintf, buffer, base_buffer_size, "%*x", 20, 305441741);
  printing_check("            edcb5433", "==", invokable::sprintf, buffer, base_buffer_size, "%*x", 20, 3989525555U);
  printing_check("            1234ABCD", "==", invokable::sprintf, buffer, base_buffer_size, "%*X", 20, 305441741);
  printing_check("            EDCB5433", "==", invokable::sprintf, buffer, base_buffer_size, "%*X", 20, 3989525555U);
  printing_check("                   x", "==", invokable::sprintf, buffer, base_buffer_size, "%*c", 20,'x');
}


void testcase_width_minus_20() {
  char buffer[base_buffer_size];
  printing_check("Hello               ", "==", invokable::sprintf, buffer, base_buffer_size, "%-20s", mds("Hello"));
  printing_check("1024                ", "==", invokable::sprintf, buffer, base_buffer_size, "%-20d", 1024);
  printing_check("-1024               ", "==", invokable::sprintf, buffer, base_buffer_size, "%-20d", -1024);
  printing_check("1024                ", "==", invokable::sprintf, buffer, base_buffer_size, "%-20i", 1024);
  printing_check("-1024               ", "==", invokable::sprintf, buffer, base_buffer_size, "%-20i", -1024);
  printing_check("1024                ", "==", invokable::sprintf, buffer, base_buffer_size, "%-20u", 1024);
  printing_check("1024.1234           ", "==", invokable::sprintf, buffer, base_buffer_size, "%-20.4f", 1024.1234);
  printing_check("4294966272          ", "==", invokable::sprintf, buffer, base_buffer_size, "%-20u", 4294966272U);
  printing_check("777                 ", "==", invokable::sprintf, buffer, base_buffer_size, "%-20o", 511);
  printing_check("37777777001         ", "==", invokable::sprintf, buffer, base_buffer_size, "%-20o", 4294966785U);
  printing_check("1234abcd            ", "==", invokable::sprintf, buffer, base_buffer_size, "%-20x", 305441741);
  printing_check("edcb5433            ", "==", invokable::sprintf, buffer, base_buffer_size, "%-20x", 3989525555U);
  printing_check("1234ABCD            ", "==", invokable::sprintf, buffer, base_buffer_size, "%-20X", 305441741);
  printing_check("EDCB5433            ", "==", invokable::sprintf, buffer, base_buffer_size, "%-20X", 3989525555U);
  printing_check("x                   ", "==", invokable::sprintf, buffer, base_buffer_size, "%-20c", 'x');
  printing_check("|    9| |9 | |    9|", "==", invokable::sprintf, buffer, base_buffer_size, "|%5d| |%-2d| |%5d|", 9, 9, 9);
  printing_check("|   10| |10| |   10|", "==", invokable::sprintf, buffer, base_buffer_size, "|%5d| |%-2d| |%5d|", 10, 10, 10);
  printing_check("|    9| |9           | |    9|", "==", invokable::sprintf, buffer, base_buffer_size, "|%5d| |%-12d| |%5d|", 9, 9, 9);
  printing_check("|   10| |10          | |   10|", "==", invokable::sprintf, buffer, base_buffer_size, "|%5d| |%-12d| |%5d|", 10, 10, 10);
}

#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
void testcase_width_0_minus_20() {
  char buffer[base_buffer_size];
  printing_check("Hello               ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-20s", mds("Hello"));
  printing_check("1024                ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-20d", 1024);
  printing_check("-1024               ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-20d", -1024);
  printing_check("1024                ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-20i", 1024);
  printing_check("-1024               ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-20i", -1024);
  printing_check("1024                ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-20u", 1024);
  printing_check("4294966272          ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-20u", 4294966272U);
  printing_check("777                 ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-20o", 511);
  printing_check("37777777001         ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-20o", 4294966785U);
  printing_check("1234abcd            ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-20x", 305441741);
  printing_check("edcb5433            ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-20x", 3989525555U);
  printing_check("1234ABCD            ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-20X", 305441741);
  printing_check("EDCB5433            ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-20X", 3989525555U);
  printing_check("x                   ", "==", invokable::sprintf, buffer, base_buffer_size, "%0-20c", 'x');
}
#endif

void testcase_padding_20() {
  char buffer[base_buffer_size];
  printing_check("00000000000000001024", "==", invokable::sprintf, buffer, base_buffer_size, "%020d", 1024);
  printing_check("-0000000000000001024", "==", invokable::sprintf, buffer, base_buffer_size, "%020d", -1024);
  printing_check("00000000000000001024", "==", invokable::sprintf, buffer, base_buffer_size, "%020i", 1024);
  printing_check("-0000000000000001024", "==", invokable::sprintf, buffer, base_buffer_size, "%020i", -1024);
  printing_check("00000000000000001024", "==", invokable::sprintf, buffer, base_buffer_size, "%020u", 1024);
  printing_check("00000000004294966272", "==", invokable::sprintf, buffer, base_buffer_size, "%020u", 4294966272U);
  printing_check("00000000000000000777", "==", invokable::sprintf, buffer, base_buffer_size, "%020o", 511);
  printing_check("00000000037777777001", "==", invokable::sprintf, buffer, base_buffer_size, "%020o", 4294966785U);
  printing_check("0000000000001234abcd", "==", invokable::sprintf, buffer, base_buffer_size, "%020x", 305441741);
  printing_check("000000000000edcb5433", "==", invokable::sprintf, buffer, base_buffer_size, "%020x", 3989525555U);
  printing_check("0000000000001234ABCD", "==", invokable::sprintf, buffer, base_buffer_size, "%020X", 305441741);
  printing_check("000000000000EDCB5433", "==", invokable::sprintf, buffer, base_buffer_size, "%020X", 3989525555U);
}


void testcase_padding_dot_20() {
  char buffer[base_buffer_size];
  printing_check("00000000000000001024", "==", invokable::sprintf, buffer, base_buffer_size, "%.20d", 1024);
  printing_check("-00000000000000001024", "==", invokable::sprintf, buffer, base_buffer_size, "%.20d", -1024);
  printing_check("00000000000000001024", "==", invokable::sprintf, buffer, base_buffer_size, "%.20i", 1024);
  printing_check("-00000000000000001024", "==", invokable::sprintf, buffer, base_buffer_size, "%.20i", -1024);
  printing_check("00000000000000001024", "==", invokable::sprintf, buffer, base_buffer_size, "%.20u", 1024);
  printing_check("00000000004294966272", "==", invokable::sprintf, buffer, base_buffer_size, "%.20u", 4294966272U);
  printing_check("00000000000000000777", "==", invokable::sprintf, buffer, base_buffer_size, "%.20o", 511);
  printing_check("00000000037777777001", "==", invokable::sprintf, buffer, base_buffer_size, "%.20o", 4294966785U);
  printing_check("0000000000001234abcd", "==", invokable::sprintf, buffer, base_buffer_size, "%.20x", 305441741);
  printing_check("000000000000edcb5433", "==", invokable::sprintf, buffer, base_buffer_size, "%.20x", 3989525555U);
  printing_check("0000000000001234ABCD", "==", invokable::sprintf, buffer, base_buffer_size, "%.20X", 305441741);
  printing_check("000000000000EDCB5433", "==", invokable::sprintf, buffer, base_buffer_size, "%.20X", 3989525555U);
}

#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
void testcase_padding_hash_020__non_standard_format() {
  char buffer[base_buffer_size];
  printing_check("00000000000000001024", "==", invokable::sprintf, buffer, base_buffer_size, "%#020d", 1024);
  printing_check("-0000000000000001024", "==", invokable::sprintf, buffer, base_buffer_size, "%#020d", -1024);
  printing_check("00000000000000001024", "==", invokable::sprintf, buffer, base_buffer_size, "%#020i", 1024);
  printing_check("-0000000000000001024", "==", invokable::sprintf, buffer, base_buffer_size, "%#020i", -1024);
  printing_check("00000000000000001024", "==", invokable::sprintf, buffer, base_buffer_size, "%#020u", 1024);
  printing_check("00000000004294966272", "==", invokable::sprintf, buffer, base_buffer_size, "%#020u", 4294966272U);
}
#endif

void testcase_padding_hash_020() {
  char buffer[base_buffer_size];
  printing_check("00000000000000000777", "==", invokable::sprintf, buffer, base_buffer_size, "%#020o", 511);
  printing_check("00000000037777777001", "==", invokable::sprintf, buffer, base_buffer_size, "%#020o", 4294966785U);
  printing_check("0x00000000001234abcd", "==", invokable::sprintf, buffer, base_buffer_size, "%#020x", 305441741);
  printing_check("0x0000000000edcb5433", "==", invokable::sprintf, buffer, base_buffer_size, "%#020x", 3989525555U);
  printing_check("0X00000000001234ABCD", "==", invokable::sprintf, buffer, base_buffer_size, "%#020X", 305441741);
  printing_check("0X0000000000EDCB5433", "==", invokable::sprintf, buffer, base_buffer_size, "%#020X", 3989525555U);
}


#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
void testcase_padding_hash_20__non_standard_format() {
  char buffer[base_buffer_size];
  printing_check("                1024", "==", invokable::sprintf, buffer, base_buffer_size, "%#20d", 1024);
  printing_check("               -1024", "==", invokable::sprintf, buffer, base_buffer_size, "%#20d", -1024);
  printing_check("                1024", "==", invokable::sprintf, buffer, base_buffer_size, "%#20i", 1024);
  printing_check("               -1024", "==", invokable::sprintf, buffer, base_buffer_size, "%#20i", -1024);
  printing_check("                1024", "==", invokable::sprintf, buffer, base_buffer_size, "%#20u", 1024);
  printing_check("          4294966272", "==", invokable::sprintf, buffer, base_buffer_size, "%#20u", 4294966272U);
}
#endif

void testcase_padding_hash_20() {
  char buffer[base_buffer_size];
  printing_check("                0777", "==", invokable::sprintf, buffer, base_buffer_size, "%#20o", 511);
  printing_check("        037777777001", "==", invokable::sprintf, buffer, base_buffer_size, "%#20o", 4294966785U);
  printing_check("          0x1234abcd", "==", invokable::sprintf, buffer, base_buffer_size, "%#20x", 305441741);
  printing_check("          0xedcb5433", "==", invokable::sprintf, buffer, base_buffer_size, "%#20x", 3989525555U);
  printing_check("          0X1234ABCD", "==", invokable::sprintf, buffer, base_buffer_size, "%#20X", 305441741);
  printing_check("          0XEDCB5433", "==", invokable::sprintf, buffer, base_buffer_size, "%#20X", 3989525555U);
}


void testcase_padding_20_dot_5() {
  char buffer[base_buffer_size];
  printing_check("               01024", "==", invokable::sprintf, buffer, base_buffer_size, "%20.5d", 1024);
  printing_check("              -01024", "==", invokable::sprintf, buffer, base_buffer_size, "%20.5d", -1024);
  printing_check("               01024", "==", invokable::sprintf, buffer, base_buffer_size, "%20.5i", 1024);
  printing_check("              -01024", "==", invokable::sprintf, buffer, base_buffer_size, "%20.5i", -1024);
  printing_check("               01024", "==", invokable::sprintf, buffer, base_buffer_size, "%20.5u", 1024);
  printing_check("          4294966272", "==", invokable::sprintf, buffer, base_buffer_size, "%20.5u", 4294966272U);
  printing_check("               00777", "==", invokable::sprintf, buffer, base_buffer_size, "%20.5o", 511);
  printing_check("         37777777001", "==", invokable::sprintf, buffer, base_buffer_size, "%20.5o", 4294966785U);
  printing_check("            1234abcd", "==", invokable::sprintf, buffer, base_buffer_size, "%20.5x", 305441741);
  printing_check("          00edcb5433", "==", invokable::sprintf, buffer, base_buffer_size, "%20.10x", 3989525555U);
  printing_check("            1234ABCD", "==", invokable::sprintf, buffer, base_buffer_size, "%20.5X", 305441741);
  printing_check("          00EDCB5433", "==", invokable::sprintf, buffer, base_buffer_size, "%20.10X", 3989525555U);
}


void testcase_padding_neg_numbers() {
  char buffer[base_buffer_size];

  // space padding
  printing_check("-5", "==", invokable::sprintf, buffer, base_buffer_size, "% 1d", -5);
  printing_check("-5", "==", invokable::sprintf, buffer, base_buffer_size, "% 2d", -5);
  printing_check(" -5", "==", invokable::sprintf, buffer, base_buffer_size, "% 3d", -5);
  printing_check("  -5", "==", invokable::sprintf, buffer, base_buffer_size, "% 4d", -5);

  // zero padding
  printing_check("-5", "==", invokable::sprintf, buffer, base_buffer_size, "%01d", -5);
  printing_check("-5", "==", invokable::sprintf, buffer, base_buffer_size, "%02d", -5);
  printing_check("-05", "==", invokable::sprintf, buffer, base_buffer_size, "%03d", -5);
  printing_check("-005", "==", invokable::sprintf, buffer, base_buffer_size, "%04d", -5);
}


void testcase_float_padding_neg_numbers() {
  char buffer[base_buffer_size];
/*
  // space padding
  printing_check("-5.0", "==", invokable::sprintf, buffer, base_buffer_size, "% 3.1f", -5.);
  printing_check("-5.0", "==", invokable::sprintf, buffer, base_buffer_size, "% 4.1f", -5.);
  printing_check(" -5.0", "==", invokable::sprintf, buffer, base_buffer_size, "% 5.1f", -5.);
*/
#ifndef PRINTF_DISABLE_SUPPORT_EXPONENTIAL
  printing_check("    -5", "==", invokable::sprintf, buffer, base_buffer_size, "% 6.1g", -5.);
/*  printing_check("-5.0e+00", "==", invokable::sprintf, buffer, base_buffer_size, "% 6.1e", -5.);
  printing_check("  -5.0e+00", "==", invokable::sprintf, buffer, base_buffer_size, "% 10.1e", -5.);
*/
#endif
/*
  // zero padding
  printing_check("-5.0", "==", invokable::sprintf, buffer, base_buffer_size, "%03.1f", -5.);
  printing_check("-5.0", "==", invokable::sprintf, buffer, base_buffer_size, "%04.1f", -5.);
  printing_check("-05.0", "==", invokable::sprintf, buffer, base_buffer_size, "%05.1f", -5.);

  // zero padding no decimal point
  printing_check("-5", "==", invokable::sprintf, buffer, base_buffer_size, "%01.0f", -5.);
  printing_check("-5", "==", invokable::sprintf, buffer, base_buffer_size, "%02.0f", -5.);
  printing_check("-05", "==", invokable::sprintf, buffer, base_buffer_size, "%03.0f", -5.);

#ifndef PRINTF_DISABLE_SUPPORT_EXPONENTIAL
  printing_check("-005.0e+00", "==", invokable::sprintf, buffer, base_buffer_size, "%010.1e", -5.);
  printing_check("-05E+00", "==", invokable::sprintf, buffer, base_buffer_size, "%07.0E", -5.);
  printing_check("-05", "==", invokable::sprintf, buffer, base_buffer_size, "%03.0g", -5.);
#endif
 */
}

void testcase_length() {
  char buffer[base_buffer_size];
  printing_check("", "==", invokable::sprintf, buffer, base_buffer_size, "%.0s", mds("Hello testing"));
  printing_check("                    ", "==", invokable::sprintf, buffer, base_buffer_size, "%20.0s", mds("Hello testing"));
  printing_check("", "==", invokable::sprintf, buffer, base_buffer_size, "%.s", mds("Hello testing"));
  printing_check("                    ", "==", invokable::sprintf, buffer, base_buffer_size, "%20.s", mds("Hello testing"));
  printing_check("                1024", "==", invokable::sprintf, buffer, base_buffer_size, "%20.0d", 1024);
  printing_check("               -1024", "==", invokable::sprintf, buffer, base_buffer_size, "%20.0d", -1024);
  printing_check("                    ", "==", invokable::sprintf, buffer, base_buffer_size, "%20.d", 0);
  printing_check("                1024", "==", invokable::sprintf, buffer, base_buffer_size, "%20.0i", 1024);
  printing_check("               -1024", "==", invokable::sprintf, buffer, base_buffer_size, "%20.i", -1024);
  printing_check("                    ", "==", invokable::sprintf, buffer, base_buffer_size, "%20.i", 0);
  printing_check("                1024", "==", invokable::sprintf, buffer, base_buffer_size, "%20.u", 1024);
  printing_check("          4294966272", "==", invokable::sprintf, buffer, base_buffer_size, "%20.0u", 4294966272U);
  printing_check("                    ", "==", invokable::sprintf, buffer, base_buffer_size, "%20.u", 0U);
  printing_check("                 777", "==", invokable::sprintf, buffer, base_buffer_size, "%20.o", 511);
  printing_check("         37777777001", "==", invokable::sprintf, buffer, base_buffer_size, "%20.0o", 4294966785U);
  printing_check("                    ", "==", invokable::sprintf, buffer, base_buffer_size, "%20.o", 0U);
  printing_check("            1234abcd", "==", invokable::sprintf, buffer, base_buffer_size, "%20.x", 305441741);
  printing_check("                                          1234abcd", "==", invokable::sprintf, buffer, base_buffer_size, "%50.x", 305441741);
  printing_check("                                          1234abcd     12345", "==", invokable::sprintf, buffer, base_buffer_size, "%50.x%10.u", 305441741, 12345);
  printing_check("            edcb5433", "==", invokable::sprintf, buffer, base_buffer_size, "%20.0x", 3989525555U);
  printing_check("                    ", "==", invokable::sprintf, buffer, base_buffer_size, "%20.x", 0U);
  printing_check("            1234ABCD", "==", invokable::sprintf, buffer, base_buffer_size, "%20.X", 305441741);
  printing_check("            EDCB5433", "==", invokable::sprintf, buffer, base_buffer_size, "%20.0X", 3989525555U);
  printing_check("                    ", "==", invokable::sprintf, buffer, base_buffer_size, "%20.X", 0U);
}

#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
void testcase_length__non_standard_format() {
  char buffer[base_buffer_size];
  printing_check("  ", "==", invokable::sprintf, buffer, base_buffer_size, "%02.0u", 0U);
  printing_check("  ", "==", invokable::sprintf, buffer, base_buffer_size, "%02.0d", 0);
}
#endif


void testcase_float() {
  char buffer[base_buffer_size];

  // test special-case floats using math.h macros
  printing_check("     nan", "==", invokable::sprintf, buffer, base_buffer_size, "%8f", (double) NAN);
  printing_check("     inf", "==", invokable::sprintf, buffer, base_buffer_size, "%8f", (double) INFINITY);
  printing_check("-inf    ", "==", invokable::sprintf, buffer, base_buffer_size, "%-8f", (double) -INFINITY);

#ifndef PRINTF_DISABLE_SUPPORT_EXPONENTIAL
  printing_check("    +inf", "==", invokable::sprintf, buffer, base_buffer_size, "%+8e", (double) INFINITY);
#endif
  printing_check("3.1415", "==", invokable::sprintf, buffer, base_buffer_size, "%.4f", 3.1415354);
  printing_check("30343.142", "==", invokable::sprintf, buffer, base_buffer_size, "%.3f", 30343.1415354);

  // switch from decimal to exponential representation
  //
  if (PRINTF_MAX_INTEGRAL_DIGITS_FOR_DECIMAL < 3) {
    printing_check("1e+3", "==", invokable::sprintf, buffer, base_buffer_size, "%.0f", (double) ((int64_t)1 * 1000 ) );
  }
  else {
    printing_check("1000", "==", invokable::sprintf, buffer, base_buffer_size, "%.0f", (double) ((int64_t)1 * 1000 ) );
  }

  if (PRINTF_MAX_INTEGRAL_DIGITS_FOR_DECIMAL < 6) {
    printing_check("1e+6", "==", invokable::sprintf, buffer, base_buffer_size, "%.0f", (double) ((int64_t)1 * 1000 * 1000 ) );
  }
  else {
    printing_check("1000000", "==", invokable::sprintf, buffer, base_buffer_size, "%.0f", (double) ((int64_t)1 * 1000 * 1000 ) );
  }

  if (PRINTF_MAX_INTEGRAL_DIGITS_FOR_DECIMAL < 9) {
    printing_check("1e+9", "==", invokable::sprintf, buffer, base_buffer_size, "%.0f", (double) ((int64_t)1 * 1000 * 1000 * 1000 ) );
  }
  else {
    printing_check("1000000000", "==", invokable::sprintf, buffer, base_buffer_size, "%.0f", (double) ((int64_t)1 * 1000 * 1000 * 1000 ) );
  }

  if (PRINTF_MAX_INTEGRAL_DIGITS_FOR_DECIMAL < 12) {
    printing_check("1e+12", "==", invokable::sprintf, buffer, base_buffer_size, "%.0f", (double) ((int64_t)1 * 1000 * 1000 * 1000 * 1000) );
  }
  else {
    printing_check("1000000000000", "==", invokable::sprintf, buffer, base_buffer_size, "%.0f", (double) ((int64_t)1 * 1000 * 1000 * 1000 * 1000) );
  }

  if (PRINTF_MAX_INTEGRAL_DIGITS_FOR_DECIMAL < 15) {
    printing_check("1e+15", "==", invokable::sprintf, buffer, base_buffer_size, "%.0f", (double) ((int64_t)1 * 1000 * 1000 * 1000 * 1000 * 1000) );
  }
  else {
    printing_check("1000000000000000", "==", invokable::sprintf, buffer, base_buffer_size, "%.0f", (double) ((int64_t)1 * 1000 * 1000 * 1000 * 1000 * 1000) );
  }
  printing_check("34", "==", invokable::sprintf, buffer, base_buffer_size, "%.0f", 34.1415354);
  printing_check("1", "==", invokable::sprintf, buffer, base_buffer_size, "%.0f", 1.3);
  printing_check("2", "==", invokable::sprintf, buffer, base_buffer_size, "%.0f", 1.55);
  printing_check("1.6", "==", invokable::sprintf, buffer, base_buffer_size, "%.1f", 1.64);
  printing_check("42.90", "==", invokable::sprintf, buffer, base_buffer_size, "%.2f", 42.8952);
  printing_check("42.895200000", "==", invokable::sprintf, buffer, base_buffer_size, "%.9f", 42.8952);
  printing_check("42.8952230000", "==", invokable::sprintf, buffer, base_buffer_size, "%.10f", 42.895223);
  printing_check("42.895223123457", "==", invokable::sprintf, buffer, base_buffer_size, "%.12f", 42.89522312345678);
  printing_check("42477.371093750000000", "==", invokable::sprintf, buffer, base_buffer_size, "%020.15f", 42477.37109375);
  printing_check("42.895223876543", "==", invokable::sprintf, buffer, base_buffer_size, "%.12f", 42.89522387654321);
  printing_check(" 42.90", "==", invokable::sprintf, buffer, base_buffer_size, "%6.2f", 42.8952);
  printing_check("+42.90", "==", invokable::sprintf, buffer, base_buffer_size, "%+6.2f", 42.8952);
  printing_check("+42.9", "==", invokable::sprintf, buffer, base_buffer_size, "%+5.1f", 42.9252);
  printing_check("42.500000", "==", invokable::sprintf, buffer, base_buffer_size, "%f", 42.5);
  printing_check("42.5", "==", invokable::sprintf, buffer, base_buffer_size, "%.1f", 42.5);
  printing_check("42167.000000", "==", invokable::sprintf, buffer, base_buffer_size, "%f", 42167.0);
  printing_check("-12345.987654321", "==", invokable::sprintf, buffer, base_buffer_size, "%.9f", -12345.987654321);
  printing_check("4.0", "==", invokable::sprintf, buffer, base_buffer_size, "%.1f", 3.999);
  printing_check("4", "==", invokable::sprintf, buffer, base_buffer_size, "%.0f", 3.5);
  printing_check("4", "==", invokable::sprintf, buffer, base_buffer_size, "%.0f", 4.5);
  printing_check("3", "==", invokable::sprintf, buffer, base_buffer_size, "%.0f", 3.49);
  printing_check("3.5", "==", invokable::sprintf, buffer, base_buffer_size, "%.1f", 3.49);
  printing_check("a0.5  ", "==", invokable::sprintf, buffer, base_buffer_size, "a%-5.1f", 0.5);
  printing_check("a0.5  end", "==", invokable::sprintf, buffer, base_buffer_size, "a%-5.1fend", 0.5);

#ifndef PRINTF_DISABLE_SUPPORT_EXPONENTIAL
  printing_check("12345.7", "==", invokable::sprintf, buffer, base_buffer_size, "%G", 12345.678);
  printing_check("12345.68", "==", invokable::sprintf, buffer, base_buffer_size, "%.7G", 12345.678);
  printing_check("1.2346E+08", "==", invokable::sprintf, buffer, base_buffer_size, "%.5G", 123456789.);
  printing_check("12345", "==", invokable::sprintf, buffer, base_buffer_size, "%.6G", 12345.);
  printing_check("  +1.235e+08", "==", invokable::sprintf, buffer, base_buffer_size, "%+12.4g", 123456789.);
  printing_check("0.0012", "==", invokable::sprintf, buffer, base_buffer_size, "%.2G", 0.001234);
  printing_check(" +0.001234", "==", invokable::sprintf, buffer, base_buffer_size, "%+10.4G", 0.001234);
  printing_check("+001.234e-05", "==", invokable::sprintf, buffer, base_buffer_size, "%+012.4g", 0.00001234);
  printing_check("-1.23e-308", "==", invokable::sprintf, buffer, base_buffer_size, "%.3g", -1.2345e-308);
  printing_check("+1.230E+308", "==", invokable::sprintf, buffer, base_buffer_size, "%+.3E", 1.23e+308);
#endif

  // out of range for float: should switch to exp notation if supported, else empty
#ifndef PRINTF_DISABLE_SUPPORT_EXPONENTIAL
  printing_check("1.0e+20", "==", invokable::sprintf, buffer, base_buffer_size, "%.1f", 1E20);
#else
  printing_check("", "==", invokable::sprintf, buffer, base_buffer_size, "%.1f", 1E20);
#endif

  // brute force float
  bool fail = false;
  std::stringstream str;
  str.precision(5);
  for (float i = -100000; i < 100000; i += 1) {
   invoke_on_device(invokable::sprintf, buffer, "%.5f", (double)(i / 10000));
    str.str("");
    str << std::fixed << i / 10000;
    fail = fail || !!strcmp(buffer, str.str().c_str());
  }
  if (fail) {
    std::cerr << "sprintf(\"" << "%.5f\" (double)(i / 10000)) failed." << std::endl;
    exit(EXIT_FAILURE);
  }


#ifndef PRINTF_DISABLE_SUPPORT_EXPONENTIAL
  // brute force exp
  str.setf(std::ios::scientific, std::ios::floatfield);
  for (float i = -1e20; i < (float) 1e20; i += (float) 1e15) {
   invoke_on_device(invokable::sprintf, buffer, "%.5f", (double) i);
    str.str("");
    str << i;
    fail = fail || !!strcmp(buffer, str.str().c_str());
  }
  if (fail) {
    std::cerr << "sprintf(\"" << "%.5f\" (double) i) failed." << std::endl;
    exit(EXIT_FAILURE);
  }
#endif
}


void testcase_types() {
  char buffer[base_buffer_size];
  printing_check("0", "==", invokable::sprintf, buffer, base_buffer_size, "%i", 0);
  printing_check("1234", "==", invokable::sprintf, buffer, base_buffer_size, "%i", 1234);
  printing_check("32767", "==", invokable::sprintf, buffer, base_buffer_size, "%i", 32767);
  printing_check("-32767", "==", invokable::sprintf, buffer, base_buffer_size, "%i", -32767);
  printing_check("30", "==", invokable::sprintf, buffer, base_buffer_size, "%li", 30L);
  printing_check("-2147483647", "==", invokable::sprintf, buffer, base_buffer_size, "%li", -2147483647L);
  printing_check("2147483647", "==", invokable::sprintf, buffer, base_buffer_size, "%li", 2147483647L);
  printing_check("30", "==", invokable::sprintf, buffer, base_buffer_size, "%lli", 30LL);
  printing_check("-9223372036854775807", "==", invokable::sprintf, buffer, base_buffer_size, "%lli", -9223372036854775807LL);
  printing_check("9223372036854775807", "==", invokable::sprintf, buffer, base_buffer_size, "%lli", 9223372036854775807LL);
  printing_check("100000", "==", invokable::sprintf, buffer, base_buffer_size, "%lu", 100000L);
  printing_check("4294967295", "==", invokable::sprintf, buffer, base_buffer_size, "%lu", 0xFFFFFFFFL);
  printing_check("281474976710656", "==", invokable::sprintf, buffer, base_buffer_size, "%llu", 281474976710656LLU);
  printing_check("18446744073709551615", "==", invokable::sprintf, buffer, base_buffer_size, "%llu", 18446744073709551615LLU);
  printing_check("2147483647", "==", invokable::sprintf, buffer, base_buffer_size, "%zu", (size_t)2147483647UL);
  printing_check("2147483647", "==", invokable::sprintf, buffer, base_buffer_size, "%zd", (size_t)2147483647UL);
  printing_check("-2147483647", "==", invokable::sprintf, buffer, base_buffer_size, "%zi", (ssize_t)-2147483647L);
  printing_check("165140", "==", invokable::sprintf, buffer, base_buffer_size, "%o", 60000);
  printing_check("57060516", "==", invokable::sprintf, buffer, base_buffer_size, "%lo", 12345678L);
  printing_check("12345678", "==", invokable::sprintf, buffer, base_buffer_size, "%lx", 0x12345678L);
  printing_check("1234567891234567", "==", invokable::sprintf, buffer, base_buffer_size, "%llx", 0x1234567891234567LLU);
  printing_check("abcdefab", "==", invokable::sprintf, buffer, base_buffer_size, "%lx", 0xabcdefabL);
  printing_check("ABCDEFAB", "==", invokable::sprintf, buffer, base_buffer_size, "%lX", 0xabcdefabL);
  printing_check("v", "==", invokable::sprintf, buffer, base_buffer_size, "%c", 'v');
  printing_check("wv", "==", invokable::sprintf, buffer, base_buffer_size, "%cv", 'w');
  printing_check("A Test", "==", invokable::sprintf, buffer, base_buffer_size, "%s", mds("A Test"));
  printing_check("255", "==", invokable::sprintf, buffer, base_buffer_size, "%hhu", (unsigned char) 0xFFU);
  printing_check("4660", "==", invokable::sprintf, buffer, base_buffer_size, "%hu", (unsigned short) 0x1234u);
  printing_check("Test100 65535", "==", invokable::sprintf, buffer, base_buffer_size, "%s%hhi %hu", mds("Test"), (char) 100, (unsigned short) 0xFFFF);
  printing_check("a", "==", invokable::sprintf, buffer, base_buffer_size, "%tx", &buffer[10] - &buffer[0]);
  printing_check("-2147483647", "==", invokable::sprintf, buffer, base_buffer_size, "%ji", (intmax_t)-2147483647L);
}

#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
void testcase_types__non_standard_format() {
  char buffer[base_buffer_size];
  printing_check("1110101001100000", "==", invokable::sprintf, buffer, base_buffer_size, "%b", 60000);
  printing_check("101111000110000101001110", "==", invokable::sprintf, buffer, base_buffer_size, "%lb", 12345678L);
}
#endif

void testcase_pointer() {
  char buffer[base_buffer_size];

  if (sizeof(void*) == 4U) {
    printing_check("0x00001234", "==", invokable::sprintf, buffer, base_buffer_size, "%p", (void*)0x1234U);
  }
  else {
    printing_check("0x0000000000001234", "==", invokable::sprintf, buffer, base_buffer_size, "%p", (void*)0x1234U);
  }

  if (sizeof(void*) == 4U) {
    printing_check("0x12345678", "==", invokable::sprintf, buffer, base_buffer_size, "%p", (void*)0x12345678U);
  }
  else {
    printing_check("0x0000000012345678", "==", invokable::sprintf, buffer, base_buffer_size, "%p", (void*)0x12345678U);
  }

  if (sizeof(void*) == 4U) {
    printing_check("0x12345678-0x7edcba98", "==", invokable::sprintf, buffer, base_buffer_size, "%p-%p", (void*)0x12345678U, (void*)0x7EDCBA98U);
  }
  else {
    printing_check("0x0000000012345678-0x000000007edcba98", "==", invokable::sprintf, buffer, base_buffer_size, "%p-%p", (void*)0x12345678U, (void*)0x7EDCBA98U);
  }

  if (sizeof(uintptr_t) == sizeof(uint64_t)) {
   printing_check("0x00000000ffffffff", "==", invokable::sprintf, buffer, base_buffer_size, "%p", (void*)(uintptr_t)0xFFFFFFFFU);
  }
  else {
   printing_check("0xffffffff", "==", invokable::sprintf, buffer, base_buffer_size, "%p", (void*)(uintptr_t)0xFFFFFFFFU);
  }
  printing_check("(nil)", "==", invokable::sprintf, buffer, base_buffer_size, "%p", NULL);
}

#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
void testcase_unknown_flag__non_standard_format() {
  char buffer[base_buffer_size];
  printing_check("kmarco", "==", invokable::sprintf, buffer, base_buffer_size, "%kmarco", 42, 37);
}
#endif

void testcase_string_length() {
  char buffer[base_buffer_size];
  printing_check("This", "==", invokable::sprintf, buffer, base_buffer_size, "%.4s", mds("This is a test"));
  printing_check("test", "==", invokable::sprintf, buffer, base_buffer_size, "%.4s", mds("test"));
  printing_check("123", "==", invokable::sprintf, buffer, base_buffer_size, "%.7s", mds("123"));
  printing_check("", "==", invokable::sprintf, buffer, base_buffer_size, "%.7s", mds(""));
  printing_check("1234ab", "==", invokable::sprintf, buffer, base_buffer_size, "%.4s%.2s", mds("123456"), mds("abcdef"));
  printing_check("123", "==", invokable::sprintf, buffer, base_buffer_size, "%.*s", 3, mds("123456"));

DISABLE_WARNING_PUSH
DISABLE_WARNING_PRINTF_FORMAT_OVERFLOW
  printing_check("(null)", "==", invokable::sprintf, buffer, base_buffer_size, "%.*s", 3, (const char*) NULL);
DISABLE_WARNING_POP
}

#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
void testcase_string_length__non_standard_format() {
  char buffer[base_buffer_size];
  printing_check(".2s", "==", invokable::sprintf, buffer, base_buffer_size, "%.4.2s", mds("123456"));
}
#endif


void testcase_buffer_length() {
  char buffer[base_buffer_size];

  printing_and_ret_check(4, nullptr, "==", invokable::snprintf, nullptr, 10, "%s", mds("Test"));

  buffer[0] = (char)0xA5;
  printing_and_ret_check(4, "", "==", invokable::snprintf, buffer, (size_t) 0, "%s", mds("Test"));
  if (buffer[0] != (char)0xA5) {
    std::cerr << "snprintf snprintf(buffer, 0, \"%s\", \"Test\") modified characters, when it should not have." << std::endl;
    exit(EXIT_FAILURE);
  }
  buffer[0] = (char)0xCC;
  printing_check("", "==", invokable::snprintf, buffer, 1, "%s", mds("Test"));
  printing_check("H", "==", invokable::snprintf, buffer, 2, "%s", mds("Hello"));

DISABLE_WARNING_PUSH
DISABLE_WARNING_PRINTF_FORMAT_OVERFLOW
  printing_check("(", "==", invokable::snprintf, buffer, 2, "%s", NULL);
DISABLE_WARNING_POP
}


void testcase_ret_value() {
  char buffer[base_buffer_size];
  printing_and_ret_check( 5, "01234",     "==", invokable::snprintf, buffer,  6, "0%s", mds("1234"));
  printing_and_ret_check( 6, "01234",     "==", invokable::snprintf, buffer,  6, "0%s", mds("12345"));
  printing_and_ret_check( 8, "01234",     "==", invokable::snprintf, buffer,  6, "0%s", mds("1234567"));
DISABLE_WARNING_PUSH
DISABLE_WARNING_PRINTF_FORMAT_OVERFLOW
  printing_and_ret_check( 7, "0(nul",     "==", invokable::snprintf, buffer,  6, "0%s", (const char*) NULL);
DISABLE_WARNING_POP
  printing_and_ret_check(12, "hello, wo", "==", invokable::snprintf, buffer, 10, "hello, world");
  printing_and_ret_check( 5, "10",        "==", invokable::snprintf, buffer,  3, "%d", 10000);
}

void testcase_misc() {
  char buffer[base_buffer_size];
  printing_check("53000atest-20 bit", "==", invokable::sprintf, buffer, base_buffer_size, "%u%u%ctest%d %s", 5, 3000, 'a', -20, mds("bit"));
  printing_check("0.33", "==", invokable::sprintf, buffer, base_buffer_size, "%.*f", 2, 0.33333333);
  printing_check("1", "==", invokable::sprintf, buffer, base_buffer_size, "%.*d", -1, 1);
  printing_check("foo", "==", invokable::sprintf, buffer, base_buffer_size, "%.3s", mds("foobar"));
  printing_check(" ", "==", invokable::sprintf, buffer, base_buffer_size, "% .0d", 0);
  printing_check("     00004", "==", invokable::sprintf, buffer, base_buffer_size, "%10.5d", 4);
  printing_check("hi x", "==", invokable::sprintf, buffer, base_buffer_size, "%*sx", -3, mds("hi"));

#ifndef PRINTF_DISABLE_SUPPORT_EXPONENTIAL
  printing_check("0.33", "==", invokable::sprintf, buffer, base_buffer_size, "%.*g", 2, 0.33333333);
  printing_check("3.33e-01", "==", invokable::sprintf, buffer, base_buffer_size, "%.*e", 2, 0.33333333);
#endif
}

#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
DISABLE_WARNING_POP
#endif

int main()
{
  testcase_fctprintf();
  testcase_vfctprintf();
  testcase_snprintf();
  testcase_vsprintf();
  testcase_vsnprintf();
  testcase_simple_sprintf();
  testcase_space_flag();
#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
  testcase_space_flag__non_standard_format();
#endif
  testcase_plus_flag();
  testcase_0_flag();
  testcase_minus_flag();
#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
  testcase_minus_flag__non_standard_format();
#endif
  testcase_hash_flag();

  // FIXME: this is not set!
#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
  testcase_hash_flag__non_standard_format();
#endif
  testcase_specifier();
  testcase_width();
  std::cout << "ok" << std::endl;
  testcase_width_20();
  testcase_width_star_20();
  testcase_width_minus_20();
#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
  testcase_width_0_minus_20();
#endif
  testcase_padding_20();
  testcase_padding_dot_20();
#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
  testcase_padding_hash_020__non_standard_format();
#endif
  testcase_padding_hash_020();
#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
  testcase_padding_hash_20__non_standard_format();
#endif
  testcase_padding_hash_20();
  testcase_padding_20_dot_5();
  testcase_padding_neg_numbers();

  testcase_float_padding_neg_numbers();
  testcase_length();
#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
  testcase_length__non_standard_format();
#endif
  testcase_float();
  testcase_types();
#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
  testcase_types__non_standard_format();
#endif
  testcase_pointer();
#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
  testcase_unknown_flag__non_standard_format();
#endif
  testcase_string_length();
#ifdef TEST_WITH_NON_STANDARD_FORMAT_STRINGS
  testcase_string_length__non_standard_format();
#endif
  testcase_buffer_length();
  testcase_ret_value();
  testcase_misc();
}
